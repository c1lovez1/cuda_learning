#include "hip/hip_runtime.h"
#include "cuda_error_check.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

typedef float FLOAT;

__global__ void reduce_baseline(const int *input, int *output, size_t N)
{
    int sum = 0;
    // size_t idx = blockIdx.x*blockDim.x+threadIdx.x;
    for (size_t i = 0; i < N; i++)
    {
        sum += input[i];
    }
    *output = sum;
}

bool CheckResult(int *out, int groudtruth, int N)
{
    if (*out != groudtruth)
    {
        return false;
    }
    return true;
}

int main()
{
    FLOAT milliseconds = 0;
    const size_t N = 25600000;

    const int blockSize = 1;
    const int GridSize = 1;


    // // 使用统一内存分配
    // int *a;
    // hipMallocManaged(&a, N * sizeof(int));
    // int *out;
    // hipMallocManaged(&out, GridSize * sizeof(int));

    int *a = (int *)malloc(N * sizeof(int));
    int *d_a;
    hipMalloc((void **)&d_a, N * sizeof(int));
    int *out = (int *)malloc(GridSize * sizeof(int));
    int *d_out;
    hipMalloc((void **)&d_out, GridSize * sizeof(int));

    for (int i = 0; i < N; i++)
    {
        a[i] = 1;
    }

    FLOAT groudtruth = N * 1;

    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    reduce_baseline<<<Grid, Block>>>(d_a, d_out, N);
    // reduce_baseline<<<Grid, Block>>>(a, out, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, GridSize * sizeof(int), hipMemcpyDeviceToHost);

    // 同步设备到主机内存（必要时）
    hipDeviceSynchronize();

    bool is_right = CheckResult(out, groudtruth, GridSize);
    if (is_right)
    {
        printf("the ans is right\n");
    }
    else
    {
        printf("the ans is wrong\n");
        for (int i = 0; i < GridSize; i++)
        {
            printf("res per block : %d ", out[i]);
        }
        printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    printf("reduce_baseline latency = %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);

    // 销毁事件
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // hipFree(a);
    // hipFree(out);

    return 0;
}
