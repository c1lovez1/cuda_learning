#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


//v4: 最后一个warp不用参与__syncthreads
//latency: 0.694ms
__device__ void WarpSharedMemReduce(volatile float* smem, int tid){
    float x = smem[tid];
    if (blockDim.x >= 64) {
      x += smem[tid + 32]; __syncwarp();
      smem[tid] = x; __syncwarp();
    }
    x += smem[tid + 16]; __syncwarp();
    smem[tid] = x; __syncwarp();
    x += smem[tid + 8]; __syncwarp();
    smem[tid] = x; __syncwarp();
    x += smem[tid + 4]; __syncwarp();
    smem[tid] = x; __syncwarp();
    x += smem[tid + 2]; __syncwarp();
    smem[tid] = x; __syncwarp();
    x += smem[tid + 1]; __syncwarp();
    smem[tid] = x; __syncwarp();
}

// Note: using blockSize as a template arg can benefit from NVCC compiler optimization, 
// which is better than using blockDim.x that is known in runtime.
template<int blockSize>
__global__ void reduce_v4(float *d_in,float *d_out){
    __shared__ float smem[blockSize];
    int tid = threadIdx.x;
    int i = blockIdx.x * (blockSize * 2) + threadIdx.x;
    smem[tid] = d_in[i] + d_in[i + blockSize];
    __syncthreads();

    for (int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            smem[tid] += smem[tid + s];
        }
        __syncthreads();
    }

    if (tid < 32) {
        WarpSharedMemReduce(smem, tid);
    }

    if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
    }
}

bool CheckResult(float *out, float groudtruth, int n){
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != groudtruth) {
        return false;
    }
    return true;
}

int main(){
    float milliseconds = 0;
    const int N = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);

    float *a = (float *)malloc(N * sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    float *out = (float*)malloc((GridSize) * sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out, (GridSize) * sizeof(float));

    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
    }

    float groudtruth = N * 1.0f;

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize / 2);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v4<blockSize / 2><<<Grid,Block>>>(d_a, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d \n", GridSize, N);
    bool is_right = CheckResult(out, groudtruth, GridSize);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i = 0; i < GridSize;i++){
            printf("resPerBlock : %lf ",out[i]);
        }
        printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    printf("reduce_v4 latency = %f ms\n", milliseconds);

    // 清理CUDA事件对象
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // 释放设备内存
    hipFree(d_a);
    hipFree(d_out);

    // 释放主机内存
    free(a);
    free(out);

    return 0;
}
