#include "hip/hip_runtime.h"
#include "cuda_error_check.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

typedef float FLOAT;

__global__ void vec_add(FLOAT *X,FLOAT *Y,FLOAT *Z,int N){
    int idx = (blockDim.x*(blockIdx.y * gridDim.x+blockIdx.x)+threadIdx.x);
    if (idx<N){
        Z[idx] = X[idx]+Y[idx]; 
    }
}

void vec_add_cpu(FLOAT *X,FLOAT *Y,FLOAT *Z,int N){
    for (int i =0;i<N;i++){
        Z[i] = X[i] + Y[i];
    }
}


int main()
{
    int N = 1000000;
    int nbytes = N * sizeof(float);

    int bs = 256;

    // 手动向上取整
    int s = ceil(sqrt(N + bs - 1. / bs));
    dim3 grid(s, s);

    FLOAT *hx, *dx;
    FLOAT *hy, *dy;
    FLOAT *hz, *dz;

    CHECK_CUDA_ERROR(hipMalloc((void **)&dx, nbytes));
    hipMalloc((void **)&dy, nbytes);
    hipMalloc((void **)&dz, nbytes);

    FLOAT millisecond = 0;

    hx = (FLOAT *)malloc(nbytes);
    hy = (FLOAT *)malloc(nbytes);
    hz = (FLOAT *)malloc(nbytes);

    for (int i = 0; i < N; i++)
    {
        hx[i] = 1;
        hy[i] = 1;
    }

    hipMemcpy(dx,hx,nbytes,hipMemcpyHostToDevice);
    hipMemcpy(dy,hy,nbytes,hipMemcpyHostToDevice);

    hipEvent_t start,stop;

    hipEventCreate(&start);// 创建事件
    hipEventCreate(&stop);
    hipEventRecord(start);

    vec_add<<<grid,bs>>>(dx,dy,dz,N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&millisecond,start,stop);
    
    hipMemcpy(hz,dz,nbytes,hipMemcpyDeviceToHost);

    FLOAT *hz_cpu = (FLOAT*) malloc(nbytes);
    vec_add_cpu(hx,hy,hz_cpu,N);

    for (int i =0;i<N;i++){
        if ((fabs(hz_cpu[i]-hz[i]))>1e-6){
            printf("Result verification failed at element index %d!\n", i);
        }
    }

    printf("Result right\n");
    std::cout<<"消耗时间："<<millisecond<<"ms"<<std::endl;

    hipFree(dx);
    hipFree(dy);
    hipFree(dz);

    free(hx);
    free(hy);
    free(hz);
    free(hz_cpu);

    return 0;
}