#include "cuda_error_check.h"

void CheckCudaError (hipError_t result,const char* const func,const char* const file,int const line){
    if (result != hipSuccess){
        fprintf(stderr,"Cuda error at %s:%d code = %d(%s) \"%s\" \n",
            file,line,static_cast<unsigned int>(result),hipGetErrorName(result),func);
        hipDeviceReset;
        exit(EXIT_FAILURE);
    }
}

void CheckCuBilasStatus(hipError_t result,const char* const func,const char* const file,int const line){
    if (result != hipSuccess){
        fprintf(stderr,"Cuda error at %s:%d code = %d(%s) \"%s\"",file,line,static_cast<unsigned int>(result),hipGetErrorName(result),func);
    }
    hipDeviceReset;
    exit(EXIT_FAILURE);
}
